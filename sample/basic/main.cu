#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

#include <cudangine/buffer.hpp>
#include <cudangine/buffer.hxx>


template< typename T >
__host__ __device__ void affect(T* _arr, size_t _index, T _value)
{
    _arr[_index] = _value;
}

__global__ void kernelPlusEquals(int* const _vecA, const int* const _vecB, unsigned _size)
{
    const unsigned globalSize = gridDim.x * blockDim.x;
    const double localSize = _size/__uint2double_rn(globalSize);
    const unsigned index = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned begin = index * localSize;
    const unsigned end = index * localSize + localSize;

    printf("range: %d, %d\n", begin, end);

    for(int i = begin ; i < end ; ++i)
    {
        affect(_vecA, i, _vecA[i] + _vecB[i]);
    }
}

int main(int argc, char **argv)
{
    const unsigned size = 35;
    int vecA[size];
    int vecB[size];

    for (int i = 0; i < size ; ++i)
    {
        affect(vecA,i,i);
        affect(vecB,i,i);
    }

    cudangine::Buffer<int> bufVecA(size, vecA);
    bufVecA.synchronizeDevice();

    cudangine::Buffer<int> bufVecB(size, vecB);
    bufVecA.synchronizeDevice();

    kernelPlusEquals<<<2,4>>>(bufVecA, bufVecB, size);
    hipDeviceSynchronize();

    bufVecA.synchronizeHost();

    for(int val : bufVecA.data())
    {
        std::cout << "result: " << val << std::endl;
    }

    return EXIT_SUCCESS;
}
